#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: 22/04/2012
 *      Author: Juan Antonio Aldea Armenteros
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define pix_per_thread 1
#define DEBUG

extern "C" {
#include "ppm.h"
}

__global__ void render(unsigned char *out, int width, int height, int max_iterations) {
    unsigned int x_dim = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_dim = blockIdx.y * blockDim.y + threadIdx.y;
    int index = 3 * width * y_dim + x_dim * 3;
    float x_origin = ((float) x_dim / width) * 3.25 - 2;
    float y_origin = ((float) y_dim / height) * 2.5 - 1.25;
    float x = 0.0;
    float y = 0.0;

    int iteration = 0;
    while (x * x + y * y <= 4 && iteration < max_iterations) {
        float xtemp = x * x - y * y + x_origin;
        y = 2 * x * y + y_origin;
        x = xtemp;
        iteration++;
    }
    //out[index]++;
    if (iteration == max_iterations) {
        out[index + 0] = 0;
        out[index + 1] = 0;
        out[index + 2] = 0;
    } else {
        out[index + 0] = iteration < 255 ? iteration : 255;
        out[index + 1] = iteration < 255 ? iteration : 255;
        out[index + 2] = iteration < 255 ? iteration : 255;
    }
}

void runCUDA(int width, int height, int max_iterations) {
    size_t buffer_size = sizeof(unsigned char) * width * height * 3;
    unsigned char *device_memory, *host_memory;
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(width / (blockDim.x), height / (2 * blockDim.y), 1);
    hipError_t cuda_error;
    hipDeviceReset();
    cuda_error = hipSetDeviceFlags(hipDeviceMapHost);
    printf("Set device: %s\n", hipGetErrorString(cuda_error));

    int host_alloc = 1;

    if (cuda_error == hipSuccess && host_alloc) {
        cuda_error = hipHostAlloc((void**) &host_memory, buffer_size,
                hipHostMallocMapped);
        printf("Host1 %s\n", hipGetErrorString(cuda_error));
        cuda_error = hipHostGetDevicePointer(&device_memory, host_memory, 0);
        printf("Host2 %s\n", hipGetErrorString(cuda_error));
#ifdef DEBUG
        cuda_error = hipMemset(device_memory, 255, buffer_size);
        printf("Host3 %s\n", hipGetErrorString(cuda_error));
#endif
    } else {
        cuda_error = hipMalloc((void **) &device_memory, buffer_size);
        printf("Device %s\n", hipGetErrorString(cuda_error));
#ifdef DEBUG
        cuda_error = hipMemset(device_memory, 255, buffer_size);
        printf("Device %s\n", hipGetErrorString(cuda_error));
#endif
        host_memory = (unsigned char *) malloc(buffer_size);
    }

    /************************************************************************/
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    render<<< gridDim, blockDim, 0 >>>(device_memory, width, height, max_iterations);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed time %f ms \n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    /*****************************************************************************/

    if (cuda_error == hipSuccess && host_alloc) {
        cuda_error = hipDeviceSynchronize();
        printf("Host barrier %s\n", hipGetErrorString(cuda_error));
    } else {
        cuda_error = hipMemcpy(host_memory, device_memory, buffer_size, hipMemcpyDeviceToHost);
        printf("Device %s\n", hipGetErrorString(cuda_error));
    }

    char path[100];
    sprintf(path, "cuda_%d_%d.ppm", height, max_iterations);
    write_ppm(path, height, width, 255, host_memory);

    if (cuda_error == hipSuccess && host_alloc) {
        cuda_error = hipHostFree(host_memory);
        printf("Host %s\n", hipGetErrorString(cuda_error));
    } else {
        cuda_error = hipFree(device_memory);
        printf("%s\n", hipGetErrorString(cuda_error));
        free(host_memory);
    }
}

int main(int argc, const char * argv[]) {
    int dim = atoi(argv[1]);
    int max_iterations = atoi(argv[2]);
    runCUDA(dim, dim, max_iterations);
    return 0;
}
